#include "hip/hip_runtime.h"
#include "set.cuh"

template<typename T>
static __global__ void k_set(const T * x, const T * y, T * dst, const int64_t ne,
        const int64_t ne10, const int64_t ne11, const int64_t ne12, const int64_t ne13,
        const int64_t s11, const int64_t s12, const int64_t s13, const int64_t offset) {

    const int64_t i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= ne) {
        return;
    }

    int64_t src1_idx = i - offset;

    int64_t tmp = src1_idx;
    const int64_t i13 = tmp / s13;
    tmp -= i13 * s13;
    const int64_t i12 = tmp / s12;
    tmp -= i12 * s12;
    const int64_t i11 = tmp / s11;
    tmp -= i11 * s11;
    const int64_t i10 = tmp;

    T val = x[i];
    if (src1_idx >= 0 && i10 < ne10 && i11 < ne11 && i12 < ne12 && i13 < ne13) {
        val = y[((i13*ne12 + i12) * ne11 + i11) * ne10 + i10];
    }
    dst[i] = val;
}

template<typename T>
static void set_cuda_impl(const T * x, const T * y, T * dst, const int64_t n_elements,
        const int64_t ne10, const int64_t ne11, const int64_t ne12, const int64_t ne13,
        const int64_t s1, const int64_t s2, const int64_t s3, const int64_t offset, hipStream_t stream) {

    const int num_blocks = (n_elements + CUDA_SET_BLOCK_SIZE - 1) / CUDA_SET_BLOCK_SIZE;

    k_set<T><<<num_blocks, CUDA_SET_BLOCK_SIZE, 0, stream>>>(x, y, dst, n_elements,
                                                             ne10, ne11, ne12, ne13,
                                                             s1, s2, s3, offset);

}

void ggml_cuda_op_set(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {


    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];

    const float * src0_d = (const float *) src0->data;
    const float * src1_d = (const float *) src1->data;
    float       * dst_d  = (float       *)  dst->data;

    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_I32);
    GGML_ASSERT(src1->type == src0->type);
    GGML_ASSERT( dst->type == src0->type);

    GGML_ASSERT(ggml_is_contiguous(src1));
    GGML_ASSERT(dst->nb[0] == ggml_element_size(dst));
    GGML_ASSERT(ggml_is_contiguously_allocated(dst));

    const int64_t s1     = dst->op_params[0] / ggml_element_size(dst);
    const int64_t s2     = dst->op_params[1] / ggml_element_size(dst);
    const int64_t s3     = dst->op_params[2] / ggml_element_size(dst);
    const int64_t offset = dst->op_params[3] / ggml_element_size(dst);
    const bool   inplace = (bool) dst->op_params[4];


    // If not inplace, copy src0 to dst first
    if (!inplace) {

        CUDA_CHECK(hipMemcpyAsync(dst->data, src0->data, ggml_nbytes(dst), hipMemcpyDeviceToDevice, stream));
    }

    const int64_t n = ggml_nelements(dst);
  

    switch (src0->type) {
        case GGML_TYPE_F32:
            set_cuda_impl<float>(src0_d, src1_d, dst_d, ggml_nelements(dst),
                                 src1->ne[0], src1->ne[1], src1->ne[2], src1->ne[3],
                                 s1, s2, s3, offset, stream);
            break;
        case GGML_TYPE_I32:
            set_cuda_impl<int32_t>((const int32_t*)src0_d, (const int32_t*)src1_d, (int32_t*)dst_d, ggml_nelements(dst),
                                   src1->ne[0], src1->ne[1], src1->ne[2], src1->ne[3],
                                   s1, s2, s3, offset, stream);
            break;
        default:
            GGML_ABORT("ggml_cuda_op_set: unsupported src0 type %s", ggml_type_name(src0->type));
            break;
    }
}
